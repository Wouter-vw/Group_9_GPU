#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Alloc.h"
#include "Particles.h"

/** allocate particle arrays */
void particle_allocate(struct parameters *param, struct particles *part,
                       int is) {
    // set species ID
    part->species_ID = is;
    // number of particles
    part->nop = param->np[is];
    // maximum number of particles
    part->npmax = param->npMax[is];

    // choose a different number of mover iterations for ions and electrons
    if (param->qom[is] < 0) {
        // electrons
        part->NiterMover = param->NiterMover;
        part->n_sub_cycles = param->n_sub_cycles;
    } else {
        // ions: only one iteration
        part->NiterMover = 1;
        part->n_sub_cycles = 1;
    }

    // particles per cell
    part->npcelx = param->npcelx[is];
    part->npcely = param->npcely[is];
    part->npcelz = param->npcelz[is];
    part->npcel = part->npcelx * part->npcely * part->npcelz;

    // cast it to required precision
    part->qom = (FPpart) param->qom[is];

    long npmax = part->npmax;

    // initialize drift and thermal velocities
    // drift
    part->u0 = (FPpart) param->u0[is];
    part->v0 = (FPpart) param->v0[is];
    part->w0 = (FPpart) param->w0[is];
    // thermal
    part->uth = (FPpart) param->uth[is];
    part->vth = (FPpart) param->vth[is];
    part->wth = (FPpart) param->wth[is];

    //////////////////////////////
    /// ALLOCATION PARTICLE ARRAYS
    //////////////////////////////
    part->data = new Particle[npmax];
}

/** deallocate */
void particle_deallocate(struct particles *part) {
    // deallocate particle variables
    delete[] part->data;
}

int particleUpdate(int i, struct particles *part, struct EMfield *field,
                   struct grid *grd, struct parameters *param) {
    // print species and subcycling
    // std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " -
    // species " << part->species_ID << " ***" << std::endl;

    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt / ((double) part->n_sub_cycles);
    FPpart dto2 = .5 * dt_sub_cycling, qomdt2 = part->qom * dto2 / param->c;
    FPpart omdtsq, denom, ut, vt, wt, udotb;

    // local (to the particle) electric and magnetic field
    FPfield Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

    // interpolation densities
    int ix, iy, iz;
    FPfield weight[2][2][2];
    FPfield xi[2], eta[2], zeta[2];

    // intermediate particle position and velocity
    FPpart xptilde, yptilde, zptilde, uptilde, vptilde, wptilde;

    // start subcycling
    xptilde = part->data[i].x;
    yptilde = part->data[i].y;
    zptilde = part->data[i].z;
    // calculate the average velocity iteratively
    for (int innter = 0; innter < part->NiterMover; innter++) {
        // interpolation G-->P
        ix = 2 + int((part->data[i].x - grd->xStart) * grd->invdx);
        iy = 2 + int((part->data[i].y - grd->yStart) * grd->invdy);
        iz = 2 + int((part->data[i].z - grd->zStart) * grd->invdz);

        // calculate weights
        xi[0] = part->data[i].x - grd->nodes[ix - 1][iy][iz].x;
        eta[0] = part->data[i].y - grd->nodes[ix][iy - 1][iz].y;
        zeta[0] = part->data[i].z - grd->nodes[ix][iy][iz - 1].z;
        xi[1] = grd->nodes[ix][iy][iz].x - part->data[i].x;
        eta[1] = grd->nodes[ix][iy][iz].y - part->data[i].y;
        zeta[1] = grd->nodes[ix][iy][iz].z - part->data[i].z;
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    weight[ii][jj][kk] = xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;

        // set to zero local electric and magnetic field
        Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++) {
                    Exl += weight[ii][jj][kk] *
                            field->electricField[ix - ii][iy - jj][iz - kk].x;
                    Eyl += weight[ii][jj][kk] *
                            field->electricField[ix - ii][iy - jj][iz - kk].y;
                    Ezl += weight[ii][jj][kk] *
                            field->electricField[ix - ii][iy - jj][iz - kk].z;
                    Bxl += weight[ii][jj][kk] *
                            field->magneticField[ix - ii][iy - jj][iz - kk].x;
                    Byl += weight[ii][jj][kk] *
                            field->magneticField[ix - ii][iy - jj][iz - kk].y;
                    Bzl += weight[ii][jj][kk] *
                            field->magneticField[ix - ii][iy - jj][iz - kk].z;
                }

        // end interpolation
        omdtsq = qomdt2 * qomdt2 * (Bxl * Bxl + Byl * Byl + Bzl * Bzl);
        denom = 1.0 / (1.0 + omdtsq);
        // solve the position equation
        ut = part->data[i].u + qomdt2 * Exl;
        vt = part->data[i].v + qomdt2 * Eyl;
        wt = part->data[i].w + qomdt2 * Ezl;
        udotb = ut * Bxl + vt * Byl + wt * Bzl;
        // solve the velocity equation
        uptilde =
                (ut + qomdt2 * (vt * Bzl - wt * Byl + qomdt2 * udotb * Bxl)) * denom;
        vptilde =
                (vt + qomdt2 * (wt * Bxl - ut * Bzl + qomdt2 * udotb * Byl)) * denom;
        wptilde =
                (wt + qomdt2 * (ut * Byl - vt * Bxl + qomdt2 * udotb * Bzl)) * denom;
        // update position
        part->data[i].x = xptilde + uptilde * dto2;
        part->data[i].y = yptilde + vptilde * dto2;
        part->data[i].z = zptilde + wptilde * dto2;
    } // end of iteration
    // update the final position and velocity
    part->data[i].u = 2.0 * uptilde - part->data[i].u;
    part->data[i].v = 2.0 * vptilde - part->data[i].v;
    part->data[i].w = 2.0 * wptilde - part->data[i].w;
    part->data[i].x = xptilde + uptilde * dt_sub_cycling;
    part->data[i].y = yptilde + vptilde * dt_sub_cycling;
    part->data[i].z = zptilde + wptilde * dt_sub_cycling;

    //////////
    //////////
    ////////// BC

    // X-DIRECTION: BC particles
    if (part->data[i].x > grd->Lx) {
        if (param->PERIODICX == true) {
            // PERIODIC
            part->data[i].x = part->data[i].x - grd->Lx;
        } else {
            // REFLECTING BC
            part->data[i].u = -part->data[i].u;
            part->data[i].x = 2 * grd->Lx - part->data[i].x;
        }
    }

    if (part->data[i].x < 0) {
        if (param->PERIODICX == true) {
            // PERIODIC
            part->data[i].x = part->data[i].x + grd->Lx;
        } else {
            // REFLECTING BC
            part->data[i].u = -part->data[i].u;
            part->data[i].x = -part->data[i].x;
        }
    }

    // Y-DIRECTION: BC particles
    if (part->data[i].y > grd->Ly) {
        if (param->PERIODICY == true) {
            // PERIODIC
            part->data[i].y = part->data[i].y - grd->Ly;
        } else {
            // REFLECTING BC
            part->data[i].v = -part->data[i].v;
            part->data[i].y = 2 * grd->Ly - part->data[i].y;
        }
    }

    if (part->data[i].y < 0) {
        if (param->PERIODICY == true) {
            // PERIODIC
            part->data[i].y = part->data[i].y + grd->Ly;
        } else {
            // REFLECTING BC
            part->data[i].v = -part->data[i].v;
            part->data[i].y = -part->data[i].y;
        }
    }

    // Z-DIRECTION: BC particles
    if (part->data[i].z > grd->Lz) {
        if (param->PERIODICZ == true) {
            // PERIODIC
            part->data[i].z = part->data[i].z - grd->Lz;
        } else {
            // REFLECTING BC
            part->data[i].w = -part->data[i].w;
            part->data[i].z = 2 * grd->Lz - part->data[i].z;
        }
    }

    if (part->data[i].z < 0) {
        if (param->PERIODICZ == true) {
            // PERIODIC
            part->data[i].z = part->data[i].z + grd->Lz;
        } else {
            // REFLECTING BC
            part->data[i].w = -part->data[i].w;
            part->data[i].z = -part->data[i].z;
        }
    }

    return (0); // exit succcesfully
} // end of the mover

/** particle mover */
int mover_PC(struct particles *part, struct EMfield *field, struct grid *grd,
             struct parameters *param) {
    // print species and subcycling
    // std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " -
    // species " << part->species_ID << " ***" << std::endl;

    // start subcycling
    for (int i_sub = 0; i_sub < part->n_sub_cycles; i_sub++) {
        // move each particle with new fields
        for (int i = 0; i < part->nop; i++) {
            particleUpdate(i, part, field, grd, param);
        } // end of subcycling
    } // end of one particle

    return (0); // exit succcesfully
} // end of the mover


/**
 * Implements particleUpdate as a CUDA kernel, including subcycling.
 */
__global__ void mover_PC_kernel(struct particles *part, struct EMfield *field,
                                struct grid *grd, struct parameters *param) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= part->nop) return;

    FPpart dt_sub_cycling = (FPpart) param->dt / ((double) part->n_sub_cycles);
    FPpart dto2 = .5 * dt_sub_cycling, qomdt2 = part->qom * dto2 / param->c;

    FPpart omdtsq, denom, ut, vt, wt, udotb;

    // local (to the particle) electric and magnetic field
    FPfield Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

    // interpolation densities
    int ix, iy, iz;
    FPfield weight[2][2][2];
    FPfield xi[2], eta[2], zeta[2];

    // intermediate particle position and velocity
    FPpart xptilde, yptilde, zptilde, uptilde, vptilde, wptilde;

    for (int i_sub = 0; i_sub < part->n_sub_cycles; i_sub++) {
        xptilde = part->data[i].x;
        yptilde = part->data[i].y;
        zptilde = part->data[i].z;

        for (int inner = 0; inner < part->NiterMover; inner++) {
            // Interpolation G-->P
            ix = 2 + int((part->data[i].x - grd->xStart) * grd->invdx);
            iy = 2 + int((part->data[i].y - grd->yStart) * grd->invdy);
            iz = 2 + int((part->data[i].z - grd->zStart) * grd->invdz);

            // Check indixing
            xi[0] =
                    part->data[i].x - grd->nodes_flat[(ix - 1) * (grd->nyn * grd->nzn) +
                                                      (iy) * grd->nzn + (iz)]
                    .x;
            eta[0] =
                    part->data[i].y -
                    grd->nodes_flat[ix * grd->nyn * grd->nzn + (iy - 1) * grd->nzn + (iz)]
                    .y;
            zeta[0] =
                    part->data[i].z -
                    grd->nodes_flat[ix * grd->nyn * grd->nzn + (iy) * grd->nzn + (iz - 1)]
                    .z;

            xi[1] =
                    grd->nodes_flat[ix * grd->nyn * grd->nzn + (iy) * grd->nzn + (iz)].x -
                    part->data[i].x;
            eta[1] =
                    grd->nodes_flat[ix * grd->nyn * grd->nzn + (iy) * grd->nzn + (iz)].y -
                    part->data[i].y;
            zeta[1] =
                    grd->nodes_flat[ix * grd->nyn * grd->nzn + (iy) * grd->nzn + (iz)].z -
                    part->data[i].z;

            for (int ii = 0; ii < 2; ii++) {
                for (int jj = 0; jj < 2; jj++) {
                    for (int kk = 0; kk < 2; kk++) {
                        weight[ii][jj][kk] = xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;
                    }
                }
            }

            Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

            for (int ii = 0; ii < 2; ii++) {
                for (int jj = 0; jj < 2; jj++) {
                    for (int kk = 0; kk < 2; kk++) {
                        // Check idx index
                        int idx = (ix - ii) * grd->nyn * grd->nzn + (iy - jj) * grd->nzn +
                                  (iz - kk);
                        Exl += weight[ii][jj][kk] * field->electricField_flat[idx].x;
                        Eyl += weight[ii][jj][kk] * field->electricField_flat[idx].y;
                        Ezl += weight[ii][jj][kk] * field->electricField_flat[idx].z;
                        Bxl += weight[ii][jj][kk] * field->magneticField_flat[idx].x;
                        Byl += weight[ii][jj][kk] * field->magneticField_flat[idx].y;
                        Bzl += weight[ii][jj][kk] * field->magneticField_flat[idx].z;
                    }
                }
            }

            // Particle motion equations
            omdtsq = qomdt2 * qomdt2 * (Bxl * Bxl + Byl * Byl + Bzl * Bzl);
            denom = 1.0 / (1.0 + omdtsq);

            ut = part->data[i].u + qomdt2 * Exl;
            vt = part->data[i].v + qomdt2 * Eyl;
            wt = part->data[i].w + qomdt2 * Ezl;
            udotb = ut * Bxl + vt * Byl + wt * Bzl;

            uptilde =
                    (ut + qomdt2 * (vt * Bzl - wt * Byl + qomdt2 * udotb * Bxl)) * denom;
            vptilde =
                    (vt + qomdt2 * (wt * Bxl - ut * Bzl + qomdt2 * udotb * Byl)) * denom;
            wptilde =
                    (wt + qomdt2 * (ut * Byl - vt * Bxl + qomdt2 * udotb * Bzl)) * denom;

            part->data[i].x = xptilde + uptilde * dto2;
            part->data[i].y = yptilde + vptilde * dto2;
            part->data[i].z = zptilde + wptilde * dto2;
        }

        // Update the final position and velocity
        part->data[i].u = 2.0 * uptilde - part->data[i].u;
        part->data[i].v = 2.0 * vptilde - part->data[i].v;
        part->data[i].w = 2.0 * wptilde - part->data[i].w;
        // update position
        part->data[i].x = xptilde + uptilde * dt_sub_cycling;
        part->data[i].y = yptilde + vptilde * dt_sub_cycling;
        part->data[i].z = zptilde + wptilde * dt_sub_cycling;

        // Boundary conditions
        if (part->data[i].x > grd->Lx) {
            if (grd->PERIODICX) {
                part->data[i].x -= grd->Lx;
            } else {
                part->data[i].u = -part->data[i].u;
                part->data[i].x = 2 * grd->Lx - part->data[i].x;
            }
        }
        if (part->data[i].x < 0) {
            if (grd->PERIODICX) {
                part->data[i].x += grd->Lx;
            } else {
                part->data[i].u = -part->data[i].u;
                part->data[i].x = -part->data[i].x;
            }
        }
        if (part->data[i].y > grd->Ly) {
            if (grd->PERIODICY) {
                part->data[i].y -= grd->Ly;
            } else {
                part->data[i].v = -part->data[i].v;
                part->data[i].y = 2 * grd->Ly - part->data[i].y;
            }
        }
        if (part->data[i].y < 0) {
            if (grd->PERIODICY) {
                part->data[i].y += grd->Ly;
            } else {
                part->data[i].v = -part->data[i].v;
                part->data[i].y = -part->data[i].y;
            }
        }
        if (part->data[i].z > grd->Lz) {
            if (grd->PERIODICZ) {
                part->data[i].z -= grd->Lz;
            } else {
                part->data[i].w = -part->data[i].w;
                part->data[i].z = 2 * grd->Lz - part->data[i].z;
            }
        }
        if (part->data[i].z < 0) {
            if (grd->PERIODICZ) {
                part->data[i].z += grd->Lz;
            } else {
                part->data[i].w = -part->data[i].w;
                part->data[i].z = -part->data[i].z;
            }
        }
    }
}


/**
 * Moves particles using the GPU. This function is a wrapper around the
 * mover_PC_kernel function that handles memory allocation and copying.
 */
int mover_PC_GPU(struct particles *part, struct EMfield *field,
                 struct grid *grd, struct parameters *param) {
    // Size calculations
    int nxn = grd->nxn;
    int nyn = grd->nyn;
    int nzn = grd->nzn;

    // 1. Allocate memory for structs on device
    EMfield *d_field;
    grid *d_grd;
    parameters *d_param;

    hipMalloc(&d_field, sizeof(EMfield));
    hipMalloc(&d_grd, sizeof(grid));
    hipMalloc(&d_param, sizeof(parameters));

    // 2. Allocate memory for arrays on device
    // Grid and field arrays
    Vec3<FPpart> *d_nodes_flat;
    Vec3<FPfield> *d_electricField_flat;
    Vec3<FPfield> *d_magneticField_flat;

    hipMalloc(&d_nodes_flat, nxn * nyn * nzn * sizeof(Vec3<FPpart>));
    hipMalloc(&d_electricField_flat, nxn * nyn * nzn * sizeof(Vec3<FPfield>));
    hipMalloc(&d_magneticField_flat, nxn * nyn * nzn * sizeof(Vec3<FPfield>));

    // 3. Copy array data to device
    // flat arrays are already initialized along with non-flat counterparts
    hipMemcpy(d_nodes_flat, grd->nodes_flat,
               nxn * nyn * nzn * sizeof(Vec3<FPpart>), hipMemcpyHostToDevice);

    hipMemcpy(d_electricField_flat, field->electricField_flat,
               nxn * nyn * nzn * sizeof(Vec3<FPfield>), hipMemcpyHostToDevice);
    hipMemcpy(d_magneticField_flat, field->magneticField_flat,
               nxn * nyn * nzn * sizeof(Vec3<FPfield>), hipMemcpyHostToDevice);

    // 4. Create temporary structs with device pointers
    EMfield temp_field = *field;
    grid temp_grd = *grd;

    // 5. Update pointers in temporary structs to point to device memory

    temp_field.electricField_flat = d_electricField_flat;
    temp_field.magneticField_flat = d_magneticField_flat;
    temp_grd.nodes_flat = d_nodes_flat;

    // 6. Copy the temporary structs with device pointers to device
    hipMemcpy(d_field, &temp_field, sizeof(EMfield), hipMemcpyHostToDevice);
    hipMemcpy(d_grd, &temp_grd, sizeof(grid), hipMemcpyHostToDevice);
    hipMemcpy(d_param, param, sizeof(parameters), hipMemcpyHostToDevice);

    // 7. Launch kernel
    int threadsPerBlock = 256;
    for (int is = 0; is < param->ns; is++) {
        auto currPart = &part[is];

        int nop = currPart->nop;
        particles *d_part;
        hipMalloc(&d_part, sizeof(particles));

        Particle *d_data;
        hipMalloc(&d_data, nop * sizeof(Particle));

        hipMemcpy(d_data, currPart->data, nop * sizeof(Particle),
                   hipMemcpyHostToDevice);
        particles temp_part = *currPart;
        temp_part.data = d_data;
        hipMemcpy(d_part, &temp_part, sizeof(particles), hipMemcpyHostToDevice);

        int blocksPerGrid = (currPart->nop + threadsPerBlock - 1) / threadsPerBlock;
        mover_PC_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_part, d_field, d_grd,
                                                            d_param);
        hipDeviceSynchronize();

        // 8. Copy results back to host
        hipMemcpy(currPart->data, d_data, nop * sizeof(Particle),
                   hipMemcpyDeviceToHost);
        hipFree(d_data);
        hipFree(d_part);
    }

    // 9. Free device memory
    hipFree(d_nodes_flat);
    hipFree(d_electricField_flat);
    hipFree(d_magneticField_flat);
    hipFree(d_field);
    hipFree(d_grd);
    hipFree(d_param);

    return 0;
}

__device__ void call_weight(int ix, int iy, int iz, FPpart x, FPpart y,
                            FPpart z, FPpart u, FPpart v, FPpart w, FPpart q,
                            Vec3<FPfield> *d_nodes, FPpart d_invVOL,
                            FPpart weight[2][2][2], int d_nyn, int d_nzn) {
    FPpart xi[2], eta[2], zeta[2];

    // Compute offsets
    xi[0] = x - d_nodes[(ix - 1) * (d_nyn * d_nzn) + (iy) * d_nzn + (iz)].x;
    eta[0] = y - d_nodes[ix * d_nyn * d_nzn + (iy - 1) * d_nzn + (iz)].y;
    zeta[0] = z - d_nodes[ix * d_nyn * d_nzn + (iy) * d_nzn + (iz - 1)].z;

    int index = ix * d_nyn * d_nzn + (iy) * d_nzn + (iz);
    xi[1] = d_nodes[index].x - x;
    eta[1] = d_nodes[index].y - y;
    zeta[1] = d_nodes[index].z - z;

    // Compute weights
    for (int ii = 0; ii < 2; ii++) {
        for (int jj = 0; jj < 2; jj++) {
            for (int kk = 0; kk < 2; kk++) {
                weight[ii][jj][kk] =
                        q * xi[ii] * eta[jj] * zeta[kk] * d_invVOL * d_invVOL;
            }
        }
    }
    // printf("Weight calculation success: (%d, %d, %d)\n", ix, iy, iz);
}

__global__ void calculate_weight(Particle *d_data, Vec3<FPfield> *d_nodes,
                                 FPpart *d_weight, FPpart d_invVOL,
                                 FPpart d_xStart, FPpart d_yStart,
                                 FPpart d_zStart, FPpart d_invdx,
                                 FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                 int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart q = d_data[i].q;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // FPpart weight[2][2][2];

        FPpart xi[2], eta[2], zeta[2];

        // Compute offsets
        xi[0] = x - d_nodes[(ix - 1) * (d_nyn * d_nzn) + (iy) * d_nzn + (iz)].x;
        eta[0] = y - d_nodes[ix * d_nyn * d_nzn + (iy - 1) * d_nzn + (iz)].y;
        zeta[0] = z - d_nodes[ix * d_nyn * d_nzn + (iy) * d_nzn + (iz - 1)].z;

        int index = ix * d_nyn * d_nzn + (iy) * d_nzn + (iz);
        xi[1] = d_nodes[index].x - x;
        eta[1] = d_nodes[index].y - y;
        zeta[1] = d_nodes[index].z - z;

        // Compute weights
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    // weight[ii][jj][kk] = q * xi[ii] * eta[jj] * zeta[kk] * d_invVOL *
                    // d_invVOL;

                    int ind = 8 * i + 4 * ii + 2 * jj + kk;
                    d_weight[ind] = q * xi[ii] * eta[jj] * zeta[kk] * d_invVOL * d_invVOL;
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_rhon(Particle *d_data, FPpart *d_rhon_flat,
                                      FPpart *d_weight, FPpart d_invVOL,
                                      FPpart d_xStart, FPpart d_yStart,
                                      FPpart d_zStart, FPpart d_invdx,
                                      FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                      int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;
                    atomicAdd(&d_rhon_flat[index], d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_Jx(Particle *d_data, FPpart *d_Jx_flat,
                                    FPpart *d_weight, FPpart d_invVOL,
                                    FPpart d_xStart, FPpart d_yStart,
                                    FPpart d_zStart, FPpart d_invdx,
                                    FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                    int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart u = d_data[i].u;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_Jx_flat[index], u * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_Jy(Particle *d_data, FPpart *d_Jy_flat,
                                    FPpart *d_weight, FPpart d_invVOL,
                                    FPpart d_xStart, FPpart d_yStart,
                                    FPpart d_zStart, FPpart d_invdx,
                                    FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                    int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart v = d_data[i].v;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // std::cout << weight[0][0][0] << std::endl;
        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_Jy_flat[index], v * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_Jz(Particle *d_data, FPpart *d_Jz_flat,
                                    FPpart *d_weight, FPpart d_invVOL,
                                    FPpart d_xStart, FPpart d_yStart,
                                    FPpart d_zStart, FPpart d_invdx,
                                    FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                    int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart w = d_data[i].w;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_Jz_flat[index], w * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_pxx(Particle *d_data, FPpart *d_pxx_flat,
                                     FPpart *d_weight, FPpart d_invVOL,
                                     FPpart d_xStart, FPpart d_yStart,
                                     FPpart d_zStart, FPpart d_invdx,
                                     FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                     int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart u = d_data[i].u;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_pxx_flat[index], u * u * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_pxy(Particle *d_data, FPpart *d_pxy_flat,
                                     FPpart *d_weight, FPpart d_invVOL,
                                     FPpart d_xStart, FPpart d_yStart,
                                     FPpart d_zStart, FPpart d_invdx,
                                     FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                     int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart u = d_data[i].u;
        FPpart v = d_data[i].v;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_pxy_flat[index], u * v * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_pxz(Particle *d_data, FPpart *d_pxz_flat,
                                     FPpart *d_weight, FPpart d_invVOL,
                                     FPpart d_xStart, FPpart d_yStart,
                                     FPpart d_zStart, FPpart d_invdx,
                                     FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                     int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart u = d_data[i].u;
        FPpart w = d_data[i].w;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_pxz_flat[index], u * w * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_pyy(Particle *d_data, FPpart *d_pyy_flat,
                                     FPpart *d_weight, FPpart d_invVOL,
                                     FPpart d_xStart, FPpart d_yStart,
                                     FPpart d_zStart, FPpart d_invdx,
                                     FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                     int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart v = d_data[i].v;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_pyy_flat[index], v * v * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_pyz(Particle *d_data, FPpart *d_pyz_flat,
                                     FPpart *d_weight, FPpart d_invVOL,
                                     FPpart d_xStart, FPpart d_yStart,
                                     FPpart d_zStart, FPpart d_invdx,
                                     FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                     int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart v = d_data[i].v;
        FPpart w = d_data[i].w;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    // int index_w = i * nop + ii * 4 + jj * 2 + zz;
                    atomicAdd(&d_pyz_flat[index], v * w * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel_pzz(Particle *d_data, FPpart *d_pzz_flat,
                                     FPpart *d_weight, FPpart d_invVOL,
                                     FPpart d_xStart, FPpart d_yStart,
                                     FPpart d_zStart, FPpart d_invdx,
                                     FPpart d_invdy, FPpart d_invdz, int d_nxn,
                                     int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        // FPpart temp[2][2][2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart w = d_data[i].w;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    int ind = 8 * i + 4 * ii + 2 * jj + kk;

                    atomicAdd(&d_pzz_flat[index], w * w * d_weight[ind]);
                }
            }
        }
    }
}

__global__ void interpP2G_kernel(
    Particle *d_data, Vec3<FPfield> *d_nodes, FPpart *d_rhon_flat,
    FPpart *d_Jx_flat, FPpart *d_Jy_flat, FPpart *d_Jz_flat, FPpart *d_pxx_flat,
    FPpart *d_pxy_flat, FPpart *d_pxz_flat, FPpart *d_pyy_flat,
    FPpart *d_pyz_flat, FPpart *d_pzz_flat, FPpart d_invVOL, FPpart d_xStart,
    FPpart d_yStart, FPpart d_zStart, FPpart d_invdx, FPpart d_invdy,
    FPpart d_invdz, int d_nxn, int d_nyn, int d_nzn, int d_nop) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < d_nop) {
        // arrays needed for interpolation
        FPpart weight[2][2][2];
        // FPpart temp[2][2][2];
        FPpart xi[2], eta[2], zeta[2];

        FPpart x = d_data[i].x;
        FPpart y = d_data[i].y;
        FPpart z = d_data[i].z;
        FPpart u = d_data[i].u;
        FPpart v = d_data[i].v;
        FPpart w = d_data[i].w;
        FPpart q = d_data[i].q;

        // index of the cell
        int ix, iy, iz;

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((x - d_xStart) * d_invdx));
        iy = 2 + int(floor((y - d_yStart) * d_invdy));
        iz = 2 + int(floor((z - d_zStart) * d_invdz));

        // Check indixing
        xi[0] = x - d_nodes[(ix - 1) * (d_nyn * d_nzn) + (iy) * d_nzn + (iz)].x;
        eta[0] = y - d_nodes[ix * d_nyn * d_nzn + (iy - 1) * d_nzn + (iz)].y;
        zeta[0] = z - d_nodes[ix * d_nyn * d_nzn + (iy) * d_nzn + (iz - 1)].z;

        int index = ix * d_nyn * d_nzn + (iy) * d_nzn + (iz);
        xi[1] = d_nodes[index].x - x;
        eta[1] = d_nodes[index].y - y;
        zeta[1] = d_nodes[index].z - z;

        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    weight[ii][jj][kk] =
                            q * xi[ii] * eta[jj] * zeta[kk] * d_invVOL * d_invVOL;
                }
            }
        }

        // Atomic updates to global memory
        for (int ii = 0; ii < 2; ii++) {
            for (int jj = 0; jj < 2; jj++) {
                for (int kk = 0; kk < 2; kk++) {
                    int index =
                            (ix - ii) * (d_nyn * d_nzn) + (iy - jj) * d_nzn + (iz - kk);
                    atomicAdd(&d_rhon_flat[index], weight[ii][jj][kk]);
                    atomicAdd(&d_Jx_flat[index], u * weight[ii][jj][kk]);
                    atomicAdd(&d_Jy_flat[index], v * weight[ii][jj][kk]);
                    atomicAdd(&d_Jz_flat[index], w * weight[ii][jj][kk]);
                    atomicAdd(&d_pxx_flat[index], u * u * weight[ii][jj][kk]);
                    atomicAdd(&d_pxy_flat[index], u * v * weight[ii][jj][kk]);
                    atomicAdd(&d_pxz_flat[index], u * w * weight[ii][jj][kk]);
                    atomicAdd(&d_pyy_flat[index], v * v * weight[ii][jj][kk]);
                    atomicAdd(&d_pyz_flat[index], v * w * weight[ii][jj][kk]);
                    atomicAdd(&d_pzz_flat[index], w * w * weight[ii][jj][kk]);
                }
            }
        }
    }
}

/**
 * Variant of interpP2G_GPU using non-async memory copies. For comparison and benchmarking purposes.
 */
void interpP2G_GPU_sync(struct particles *part, struct interpDensSpecies *ids,
                        struct grid *grd) {
    // corresponds to XN, YN etc. also Ex, Bxn COPY for slimpicity
    int nxn = grd->nxn;
    int nyn = grd->nyn;
    int nzn = grd->nzn;
    int nop = part->nop;

    // Device pointers -> Free at the end
    Particle *d_data;
    FPpart *d_weight;
    Vec3<FPfield> *d_nodes;
    FPpart *d_rhon_flat, *d_Jx_flat, *d_Jy_flat, *d_Jz_flat;
    FPpart *d_pxx_flat, *d_pxy_flat, *d_pxz_flat, *d_pyy_flat, *d_pyz_flat,
            *d_pzz_flat;

    // Allocate memory on the device
    hipMalloc(&d_data, nop * sizeof(Particle));
    hipMalloc(&d_weight, 8 * nop * sizeof(FPpart));
    hipMalloc(&d_nodes, nxn * nyn * nzn * sizeof(Vec3<FPfield>));
    hipMalloc(&d_rhon_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_Jx_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_Jy_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_Jz_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pxx_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pxy_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pxz_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pyy_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pyz_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pzz_flat, nxn * nyn * nzn * sizeof(FPpart));

    int currentStreamSize = nxn * nyn * nzn * sizeof(FPpart);

    // Copy data from host to device

    hipMemcpy(d_data, part->data, nop * sizeof(Particle),
               hipMemcpyHostToDevice);
    hipMemcpy(d_nodes, grd->nodes_flat,
               nxn * nyn * nzn * sizeof(Vec3<FPfield>),
               hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (part->nop + threadsPerBlock - 1) / threadsPerBlock;

    calculate_weight<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_nodes, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);


    hipMemcpy(d_rhon_flat, ids->rhon_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jx_flat, ids->Jx_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jy_flat, ids->Jy_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jz_flat, ids->Jz_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pxx_flat, ids->pxx_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pxy_flat, ids->pxy_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pxz_flat, ids->pxz_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pyy_flat, ids->pyy_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pyz_flat, ids->pyz_flat, currentStreamSize,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pzz_flat, ids->pzz_flat, currentStreamSize,
               hipMemcpyHostToDevice);

    interpP2G_kernel_rhon<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_rhon_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_Jx<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_Jx_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_Jy<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_Jy_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_Jz<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_Jz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pxx<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_pxx_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pxy<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_pxy_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pxz<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_pxz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pyy<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_pyy_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pyz<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_pyz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pzz<<<blocksPerGrid, threadsPerBlock, 0>>>(
        d_data, d_pzz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);


    hipMemcpy(ids->rhon_flat, d_rhon_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jx_flat, d_Jx_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jy_flat, d_Jy_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jz_flat, d_Jz_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxx_flat, d_pxx_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxy_flat, d_pxy_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxz_flat, d_pxz_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyy_flat, d_pyy_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyz_flat, d_pyz_flat, currentStreamSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(ids->pzz_flat, d_pzz_flat, currentStreamSize,
               hipMemcpyDeviceToHost);

    // Free cuda arrays
    hipFree(d_data);
    hipFree(d_nodes);
    hipFree(d_rhon_flat);
    hipFree(d_Jx_flat);
    hipFree(d_Jy_flat);
    hipFree(d_Jz_flat);
    hipFree(d_pxx_flat);
    hipFree(d_pxy_flat);
    hipFree(d_pxz_flat);
    hipFree(d_pyy_flat);
    hipFree(d_pyz_flat);
    hipFree(d_pzz_flat);
    hipFree(d_weight);
}

void interpP2G_GPU(struct particles *part, struct interpDensSpecies *ids,
                   struct grid *grd) {
    int nxn = grd->nxn;
    int nyn = grd->nyn;
    int nzn = grd->nzn;
    int nop = part->nop;

    constexpr int nStreams = 10;
    hipStream_t stream[nStreams];
    for (auto &i: stream) hipStreamCreate(&i);

    // Device pointers -> Free at the end
    Particle *d_data;
    FPpart *d_weight;
    Vec3<FPfield> *d_nodes;
    FPpart *d_rhon_flat, *d_Jx_flat, *d_Jy_flat, *d_Jz_flat;
    FPpart *d_pxx_flat, *d_pxy_flat, *d_pxz_flat, *d_pyy_flat, *d_pyz_flat,
            *d_pzz_flat;

    // Allocate memory on the device
    hipMalloc(&d_data, nop * sizeof(Particle));
    hipMalloc(&d_weight, 8 * nop * sizeof(FPpart));
    hipMalloc(&d_nodes, nxn * nyn * nzn * sizeof(Vec3<FPfield>));
    hipMalloc(&d_rhon_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_Jx_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_Jy_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_Jz_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pxx_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pxy_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pxz_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pyy_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pyz_flat, nxn * nyn * nzn * sizeof(FPpart));
    hipMalloc(&d_pzz_flat, nxn * nyn * nzn * sizeof(FPpart));

    int currentStreamSize = nxn * nyn * nzn * sizeof(FPpart);

    // Copy data from host to device

    hipMemcpyAsync(d_data, part->data, nop * sizeof(Particle),
                    hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_nodes, grd->nodes_flat,
                    nxn * nyn * nzn * sizeof(Vec3<FPfield>),
                    hipMemcpyHostToDevice, stream[1]);

    for (int i = 0; i < 2; ++i) hipStreamSynchronize(stream[i]);

    int threadsPerBlock = 256;
    int blocksPerGrid = (part->nop + threadsPerBlock - 1) / threadsPerBlock;

    calculate_weight<<<blocksPerGrid, threadsPerBlock, 0, stream[5]>>>(
        d_data, d_nodes, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);

    hipMemcpyAsync(d_rhon_flat, ids->rhon_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_Jx_flat, ids->Jx_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(d_Jy_flat, ids->Jy_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[2]);
    hipMemcpyAsync(d_Jz_flat, ids->Jz_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[3]);
    hipMemcpyAsync(d_pxx_flat, ids->pxx_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[4]);
    hipMemcpyAsync(d_pxy_flat, ids->pxy_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[5]);
    hipMemcpyAsync(d_pxz_flat, ids->pxz_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[6]);
    hipMemcpyAsync(d_pyy_flat, ids->pyy_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[7]);
    hipMemcpyAsync(d_pyz_flat, ids->pyz_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[8]);
    hipMemcpyAsync(d_pzz_flat, ids->pzz_flat, currentStreamSize,
                    hipMemcpyHostToDevice, stream[9]);

    for (int i = 0; i < nStreams; ++i) hipStreamSynchronize(stream[i]);

    interpP2G_kernel_rhon<<<blocksPerGrid, threadsPerBlock, 0, stream[0]>>>(
        d_data, d_rhon_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_Jx<<<blocksPerGrid, threadsPerBlock, 0, stream[1]>>>(
        d_data, d_Jx_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_Jy<<<blocksPerGrid, threadsPerBlock, 0, stream[2]>>>(
        d_data, d_Jy_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_Jz<<<blocksPerGrid, threadsPerBlock, 0, stream[3]>>>(
        d_data, d_Jz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pxx<<<blocksPerGrid, threadsPerBlock, 0, stream[4]>>>(
        d_data, d_pxx_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pxy<<<blocksPerGrid, threadsPerBlock, 0, stream[5]>>>(
        d_data, d_pxy_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pxz<<<blocksPerGrid, threadsPerBlock, 0, stream[6]>>>(
        d_data, d_pxz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pyy<<<blocksPerGrid, threadsPerBlock, 0, stream[7]>>>(
        d_data, d_pyy_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pyz<<<blocksPerGrid, threadsPerBlock, 0, stream[8]>>>(
        d_data, d_pyz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);
    interpP2G_kernel_pzz<<<blocksPerGrid, threadsPerBlock, 0, stream[9]>>>(
        d_data, d_pzz_flat, d_weight, grd->invVOL, grd->xStart, grd->yStart,
        grd->zStart, grd->invdx, grd->invdy, grd->invdz, nxn, nyn, nzn, nop);

    hipMemcpyAsync(ids->rhon_flat, d_rhon_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[0]);
    hipMemcpyAsync(ids->Jx_flat, d_Jx_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[1]);
    hipMemcpyAsync(ids->Jy_flat, d_Jy_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[2]);
    hipMemcpyAsync(ids->Jz_flat, d_Jz_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[3]);
    hipMemcpyAsync(ids->pxx_flat, d_pxx_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[4]);
    hipMemcpyAsync(ids->pxy_flat, d_pxy_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[5]);
    hipMemcpyAsync(ids->pxz_flat, d_pxz_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[6]);
    hipMemcpyAsync(ids->pyy_flat, d_pyy_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[7]);
    hipMemcpyAsync(ids->pyz_flat, d_pyz_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[8]);
    hipMemcpyAsync(ids->pzz_flat, d_pzz_flat, currentStreamSize,
                    hipMemcpyDeviceToHost, stream[9]);

    for (int i = 0; i < nStreams; ++i) hipStreamSynchronize(stream[i]);
    // Free cuda arrays
    hipFree(d_data);
    hipFree(d_nodes);
    hipFree(d_rhon_flat);
    hipFree(d_Jx_flat);
    hipFree(d_Jy_flat);
    hipFree(d_Jz_flat);
    hipFree(d_pxx_flat);
    hipFree(d_pxy_flat);
    hipFree(d_pxz_flat);
    hipFree(d_pyy_flat);
    hipFree(d_pyz_flat);
    hipFree(d_pzz_flat);
    hipFree(d_weight);
    for (int i = 0; i < nStreams; ++i) hipStreamDestroy(stream[i]);
}


/** Interpolation Particle --> Grid: This is for species */
void interpP2G(struct particles *part, struct interpDensSpecies *ids,
               struct grid *grd) {
    // arrays needed for interpolation
    FPpart weight[2][2][2];
    FPpart temp[2][2][2];
    FPpart xi[2], eta[2], zeta[2];

    // index of the cell
    int ix, iy, iz;

    for (long long i = 0; i < part->nop; i++) {
        // determine cell: can we change to int()? is it faster?
        ix = 2 + int(floor((part->data[i].x - grd->xStart) * grd->invdx));
        iy = 2 + int(floor((part->data[i].y - grd->yStart) * grd->invdy));
        iz = 2 + int(floor((part->data[i].z - grd->zStart) * grd->invdz));

        // distances from node
        xi[0] = part->data[i].x - grd->nodes[ix - 1][iy][iz].x;
        eta[0] = part->data[i].y - grd->nodes[ix][iy - 1][iz].y;
        zeta[0] = part->data[i].z - grd->nodes[ix][iy][iz - 1].z;
        xi[1] = grd->nodes[ix][iy][iz].x - part->data[i].x;
        eta[1] = grd->nodes[ix][iy][iz].y - part->data[i].y;
        zeta[1] = grd->nodes[ix][iy][iz].z - part->data[i].z;

        // calculate the weights for different nodes
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    weight[ii][jj][kk] =
                            part->data[i].q * xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;

        //////////////////////////
        // add charge density
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->rhon[ix - ii][iy - jj][iz - kk] +=
                            weight[ii][jj][kk] * grd->invVOL;

        ////////////////////////////
        // add current density - Jx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->data[i].u * weight[ii][jj][kk];

        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        ////////////////////////////
        // add current density - Jy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->data[i].v * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        ////////////////////////////
        // add current density - Jz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->data[i].w * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->Jz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        ////////////////////////////
        // add pressure pxx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] =
                            part->data[i].u * part->data[i].u * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        ////////////////////////////
        // add pressure pxy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] =
                            part->data[i].u * part->data[i].v * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        /////////////////////////////
        // add pressure pxz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] =
                            part->data[i].u * part->data[i].w * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pxz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        /////////////////////////////
        // add pressure pyy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] =
                            part->data[i].v * part->data[i].v * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pyy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        /////////////////////////////
        // add pressure pyz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] =
                            part->data[i].v * part->data[i].w * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pyz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

        /////////////////////////////
        // add pressure pzz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] =
                            part->data[i].w * part->data[i].w * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    ids->pzz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
    }
}

// GPU Mover Time / Cycle   (s) = 0.250327
//    CPU Mover Time / Cycle   (s) = 2.89693
