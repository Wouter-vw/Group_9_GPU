#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Alloc.h"
#include "Particles.h"

/** allocate particle arrays */
void particle_allocate(struct parameters* param, struct particles* part,
                       int is) {
  // set species ID
  part->species_ID = is;
  // number of particles
  part->nop = param->np[is];
  // maximum number of particles
  part->npmax = param->npMax[is];

  // choose a different number of mover iterations for ions and electrons
  if (param->qom[is] < 0) {
    // electrons
    part->NiterMover = param->NiterMover;
    part->n_sub_cycles = param->n_sub_cycles;
  } else {
    // ions: only one iteration
    part->NiterMover = 1;
    part->n_sub_cycles = 1;
  }

  // particles per cell
  part->npcelx = param->npcelx[is];
  part->npcely = param->npcely[is];
  part->npcelz = param->npcelz[is];
  part->npcel = part->npcelx * part->npcely * part->npcelz;

  // cast it to required precision
  part->qom = (FPpart)param->qom[is];

  long npmax = part->npmax;

  // initialize drift and thermal velocities
  // drift
  part->u0 = (FPpart)param->u0[is];
  part->v0 = (FPpart)param->v0[is];
  part->w0 = (FPpart)param->w0[is];
  // thermal
  part->uth = (FPpart)param->uth[is];
  part->vth = (FPpart)param->vth[is];
  part->wth = (FPpart)param->wth[is];

  //////////////////////////////
  /// ALLOCATION PARTICLE ARRAYS
  //////////////////////////////
  part->x = new FPpart[npmax];
  part->y = new FPpart[npmax];
  part->z = new FPpart[npmax];
  // allocate velocity
  part->u = new FPpart[npmax];
  part->v = new FPpart[npmax];
  part->w = new FPpart[npmax];
  // allocate charge = q * statistical weight
  part->q = new FPinterp[npmax];
}

/** deallocate */
void particle_deallocate(struct particles* part) {
  // deallocate particle variables
  delete[] part->x;
  delete[] part->y;
  delete[] part->z;
  delete[] part->u;
  delete[] part->v;
  delete[] part->w;
  delete[] part->q;
}

int particleUpdate(int i, struct particles* part, struct EMfield* field,
                   struct grid* grd, struct parameters* param) {
  // print species and subcycling
  // std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " -
  // species " << part->species_ID << " ***" << std::endl;

  // auxiliary variables
  FPpart dt_sub_cycling = (FPpart)param->dt / ((double)part->n_sub_cycles);
  FPpart dto2 = .5 * dt_sub_cycling, qomdt2 = part->qom * dto2 / param->c;
  FPpart omdtsq, denom, ut, vt, wt, udotb;

  // local (to the particle) electric and magnetic field
  FPfield Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

  // interpolation densities
  int ix, iy, iz;
  FPfield weight[2][2][2];
  FPfield xi[2], eta[2], zeta[2];

  // intermediate particle position and velocity
  FPpart xptilde, yptilde, zptilde, uptilde, vptilde, wptilde;

  // start subcycling
  for (int i_sub = 0; i_sub < part->n_sub_cycles; i_sub++) {
    xptilde = part->x[i];
    yptilde = part->y[i];
    zptilde = part->z[i];
    // calculate the average velocity iteratively
    for (int innter = 0; innter < part->NiterMover; innter++) {
      // interpolation G-->P
      ix = 2 + int((part->x[i] - grd->xStart) * grd->invdx);
      iy = 2 + int((part->y[i] - grd->yStart) * grd->invdy);
      iz = 2 + int((part->z[i] - grd->zStart) * grd->invdz);

      // calculate weights
      xi[0] = part->x[i] - grd->XN[ix - 1][iy][iz];
      eta[0] = part->y[i] - grd->YN[ix][iy - 1][iz];
      zeta[0] = part->z[i] - grd->ZN[ix][iy][iz - 1];
      xi[1] = grd->XN[ix][iy][iz] - part->x[i];
      eta[1] = grd->YN[ix][iy][iz] - part->y[i];
      zeta[1] = grd->ZN[ix][iy][iz] - part->z[i];
      for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
          for (int kk = 0; kk < 2; kk++)
            weight[ii][jj][kk] = xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;

      // set to zero local electric and magnetic field
      Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

      for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
          for (int kk = 0; kk < 2; kk++) {
            Exl += weight[ii][jj][kk] * field->Ex[ix - ii][iy - jj][iz - kk];
            Eyl += weight[ii][jj][kk] * field->Ey[ix - ii][iy - jj][iz - kk];
            Ezl += weight[ii][jj][kk] * field->Ez[ix - ii][iy - jj][iz - kk];
            Bxl += weight[ii][jj][kk] * field->Bxn[ix - ii][iy - jj][iz - kk];
            Byl += weight[ii][jj][kk] * field->Byn[ix - ii][iy - jj][iz - kk];
            Bzl += weight[ii][jj][kk] * field->Bzn[ix - ii][iy - jj][iz - kk];
          }

      // end interpolation
      omdtsq = qomdt2 * qomdt2 * (Bxl * Bxl + Byl * Byl + Bzl * Bzl);
      denom = 1.0 / (1.0 + omdtsq);
      // solve the position equation
      ut = part->u[i] + qomdt2 * Exl;
      vt = part->v[i] + qomdt2 * Eyl;
      wt = part->w[i] + qomdt2 * Ezl;
      udotb = ut * Bxl + vt * Byl + wt * Bzl;
      // solve the velocity equation
      uptilde =
          (ut + qomdt2 * (vt * Bzl - wt * Byl + qomdt2 * udotb * Bxl)) * denom;
      vptilde =
          (vt + qomdt2 * (wt * Bxl - ut * Bzl + qomdt2 * udotb * Byl)) * denom;
      wptilde =
          (wt + qomdt2 * (ut * Byl - vt * Bxl + qomdt2 * udotb * Bzl)) * denom;
      // update position
      part->x[i] = xptilde + uptilde * dto2;
      part->y[i] = yptilde + vptilde * dto2;
      part->z[i] = zptilde + wptilde * dto2;
    }  // end of iteration
    // update the final position and velocity
    part->u[i] = 2.0 * uptilde - part->u[i];
    part->v[i] = 2.0 * vptilde - part->v[i];
    part->w[i] = 2.0 * wptilde - part->w[i];
    part->x[i] = xptilde + uptilde * dt_sub_cycling;
    part->y[i] = yptilde + vptilde * dt_sub_cycling;
    part->z[i] = zptilde + wptilde * dt_sub_cycling;

    //////////
    //////////
    ////////// BC

    // X-DIRECTION: BC particles
    if (part->x[i] > grd->Lx) {
      if (param->PERIODICX == true) {
        // PERIODIC
        part->x[i] = part->x[i] - grd->Lx;
      } else {
        // REFLECTING BC
        part->u[i] = -part->u[i];
        part->x[i] = 2 * grd->Lx - part->x[i];
      }
    }

    if (part->x[i] < 0) {
      if (param->PERIODICX == true) {
        // PERIODIC
        part->x[i] = part->x[i] + grd->Lx;
      } else {
        // REFLECTING BC
        part->u[i] = -part->u[i];
        part->x[i] = -part->x[i];
      }
    }

    // Y-DIRECTION: BC particles
    if (part->y[i] > grd->Ly) {
      if (param->PERIODICY == true) {
        // PERIODIC
        part->y[i] = part->y[i] - grd->Ly;
      } else {
        // REFLECTING BC
        part->v[i] = -part->v[i];
        part->y[i] = 2 * grd->Ly - part->y[i];
      }
    }

    if (part->y[i] < 0) {
      if (param->PERIODICY == true) {
        // PERIODIC
        part->y[i] = part->y[i] + grd->Ly;
      } else {
        // REFLECTING BC
        part->v[i] = -part->v[i];
        part->y[i] = -part->y[i];
      }
    }

    // Z-DIRECTION: BC particles
    if (part->z[i] > grd->Lz) {
      if (param->PERIODICZ == true) {
        // PERIODIC
        part->z[i] = part->z[i] - grd->Lz;
      } else {
        // REFLECTING BC
        part->w[i] = -part->w[i];
        part->z[i] = 2 * grd->Lz - part->z[i];
      }
    }

    if (part->z[i] < 0) {
      if (param->PERIODICZ == true) {
        // PERIODIC
        part->z[i] = part->z[i] + grd->Lz;
      } else {
        // REFLECTING BC
        part->w[i] = -part->w[i];
        part->z[i] = -part->z[i];
      }
    }
  }  // end of one particle

  return (0);  // exit succcesfully
}  // end of the mover

/** particle mover */
int mover_PC(struct particles* part, struct EMfield* field, struct grid* grd,
             struct parameters* param) {
  // print species and subcycling
  // std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " -
  // species " << part->species_ID << " ***" << std::endl;

  // start subcycling
  for (int i_sub = 0; i_sub < part->n_sub_cycles; i_sub++) {
    // move each particle with new fields
    for (int i = 0; i < part->nop; i++) {
      particleUpdate(i, part, field, grd, param);
    }  // end of subcycling
  }  // end of one particle

  return (0);  // exit succcesfully
}  // end of the mover

__global__ void mover_PC_kernel(
    FPpart* d_x, FPpart* d_y, FPpart* d_z, FPpart* d_u, FPpart* d_v,
    FPpart* d_w, FPpart* d_XN_flat, FPpart* d_YN_flat, FPpart* d_ZN_flat,
    FPfield* d_Ex_flat, FPfield* d_Ey_flat, FPfield* d_Ez_flat,
    FPfield* d_Bxn_flat, FPfield* d_Byn_flat, FPfield* d_Bzn_flat,
    FPpart d_invVOL, FPpart d_xStart, FPpart d_yStart, FPpart d_zStart,
    FPpart d_invdx, FPpart d_invdy, FPpart d_invdz, FPpart d_Lx, FPpart d_Ly,
    FPpart d_Lz, FPpart dt_sub_cycling, FPpart dto2, FPpart qomdt2,
    bool d_PERIODICX, bool d_PERIODICY, bool d_PERIODICZ, int d_nxn, int d_nyn,
    int d_nzn, int d_nop, int d_NiterMover) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  FPpart omdtsq, denom, ut, vt, wt, udotb;

  // local (to the particle) electric and magnetic field
  FPfield Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

  // interpolation densities
  int ix, iy, iz;
  FPfield weight[2][2][2];
  FPfield xi[2], eta[2], zeta[2];

  // intermediate particle position and velocity
  FPpart xptilde, yptilde, zptilde, uptilde, vptilde, wptilde;

  if (i < d_nop) {
    xptilde = d_x[i];
    yptilde = d_y[i];
    zptilde = d_z[i];

    for (int inner = 0; inner < d_NiterMover; inner++) {
      // Interpolation G-->P
      ix = 2 + int((d_x[i] - d_xStart) * d_invdx);
      iy = 2 + int((d_y[i] - d_yStart) * d_invdy);
      iz = 2 + int((d_z[i] - d_zStart) * d_invdz);

      // Check indixing
      xi[0] =
          d_x[i] - d_XN_flat[(ix - 1) * (d_nyn * d_nzn) + (iy)*d_nzn + (iz)];
      eta[0] = d_y[i] - d_YN_flat[ix * d_nyn * d_nzn + (iy - 1) * d_nzn + (iz)];
      zeta[0] = d_z[i] - d_ZN_flat[ix * d_nyn * d_nzn + (iy)*d_nzn + (iz - 1)];

      xi[1] = d_XN_flat[ix * d_nyn * d_nzn + (iy)*d_nzn + (iz)] - d_x[i];
      eta[1] = d_YN_flat[ix * d_nyn * d_nzn + (iy)*d_nzn + (iz)] - d_y[i];
      zeta[1] = d_ZN_flat[ix * d_nyn * d_nzn + (iy)*d_nzn + (iz)] - d_z[i];

      for (int ii = 0; ii < 2; ii++) {
        for (int jj = 0; jj < 2; jj++) {
          for (int kk = 0; kk < 2; kk++) {
            weight[ii][jj][kk] = xi[ii] * eta[jj] * zeta[kk] * d_invVOL;
          }
        }
      }

      Exl = 0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;

      for (int ii = 0; ii < 2; ii++) {
        for (int jj = 0; jj < 2; jj++) {
          for (int kk = 0; kk < 2; kk++) {
            // Check idx index
            int idx = (ix - ii) * d_nyn * d_nzn + (iy - jj) * d_nzn + (iz - kk);
            Exl += weight[ii][jj][kk] * d_Ex_flat[idx];
            Eyl += weight[ii][jj][kk] * d_Ey_flat[idx];
            Ezl += weight[ii][jj][kk] * d_Ez_flat[idx];
            Bxl += weight[ii][jj][kk] * d_Bxn_flat[idx];
            Byl += weight[ii][jj][kk] * d_Byn_flat[idx];
            Bzl += weight[ii][jj][kk] * d_Bzn_flat[idx];
          }
        }
      }

      // Particle motion equations
      omdtsq = qomdt2 * qomdt2 * (Bxl * Bxl + Byl * Byl + Bzl * Bzl);
      denom = 1.0 / (1.0 + omdtsq);

      ut = d_u[i] + qomdt2 * Exl;
      vt = d_v[i] + qomdt2 * Eyl;
      wt = d_w[i] + qomdt2 * Ezl;
      udotb = ut * Bxl + vt * Byl + wt * Bzl;

      uptilde =
          (ut + qomdt2 * (vt * Bzl - wt * Byl + qomdt2 * udotb * Bxl)) * denom;
      vptilde =
          (vt + qomdt2 * (wt * Bxl - ut * Bzl + qomdt2 * udotb * Byl)) * denom;
      wptilde =
          (wt + qomdt2 * (ut * Byl - vt * Bxl + qomdt2 * udotb * Bzl)) * denom;

      d_x[i] = xptilde + uptilde * dto2;
      d_y[i] = yptilde + vptilde * dto2;
      d_z[i] = zptilde + wptilde * dto2;
    }

    // Update the final position and velocity
    d_u[i] = 2.0 * uptilde - d_u[i];
    d_v[i] = 2.0 * vptilde - d_v[i];
    d_w[i] = 2.0 * wptilde - d_w[i];
    // update position
    d_x[i] = xptilde + uptilde * dt_sub_cycling;
    d_y[i] = yptilde + vptilde * dt_sub_cycling;
    d_z[i] = zptilde + wptilde * dt_sub_cycling;

    // Boundary conditions
    if (d_x[i] > d_Lx) {
      if (d_PERIODICX) {
        d_x[i] -= d_Lx;
      } else {
        d_u[i] = -d_u[i];
        d_x[i] = 2 * d_Lx - d_x[i];
      }
    }
    if (d_x[i] < 0) {
      if (d_PERIODICX) {
        d_x[i] += d_Lx;
      } else {
        d_u[i] = -d_u[i];
        d_x[i] = -d_x[i];
      }
    }
    if (d_y[i] > d_Ly) {
      if (d_PERIODICY) {
        d_y[i] -= d_Ly;
      } else {
        d_v[i] = -d_v[i];
        d_y[i] = 2 * d_Ly - d_y[i];
      }
    }
    if (d_y[i] < 0) {
      if (d_PERIODICY) {
        d_y[i] += d_Ly;
      } else {
        d_v[i] = -d_v[i];
        d_y[i] = -d_y[i];
      }
    }
    if (d_z[i] > d_Lz) {
      if (d_PERIODICZ) {
        d_z[i] -= d_Lz;
      } else {
        d_w[i] = -d_w[i];
        d_z[i] = 2 * d_Lz - d_z[i];
      }
    }
    if (d_z[i] < 0) {
      if (d_PERIODICZ) {
        d_z[i] += d_Lz;
      } else {
        d_w[i] = -d_w[i];
        d_z[i] = -d_z[i];
      }
    }
  }
}

int mover_PC_GPU(struct particles* part, struct EMfield* field,
                 struct grid* grd, struct parameters* param) {
  // print species and subcycling
  // std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " -
  // species " << part->species_ID << " ***" << std::endl;

  // auxiliary variables
  FPpart dt_sub_cycling = (FPpart)param->dt / ((double)part->n_sub_cycles);
  FPpart dto2 = .5 * dt_sub_cycling, qomdt2 = part->qom * dto2 / param->c;

  // corresponds to XN, YN etc. also Ex, Bxn COPY for slimpicity
  int nxn = grd->nxn;
  int nyn = grd->nyn;
  int nzn = grd->nzn;
  int nop = part->nop;

  // Device pointers -> Free at the end
  FPpart *d_x, *d_y, *d_z;
  FPpart *d_u, *d_v, *d_w;
  FPpart *d_XN_flat, *d_YN_flat, *d_ZN_flat;
  FPfield *d_Ex_flat, *d_Ey_flat, *d_Ez_flat, *d_Bxn_flat, *d_Byn_flat,
      *d_Bzn_flat;

  // Allocate memory on the device
  hipMalloc(&d_x, nop * sizeof(FPpart));
  hipMalloc(&d_y, nop * sizeof(FPpart));
  hipMalloc(&d_z, nop * sizeof(FPpart));
  hipMalloc(&d_u, nop * sizeof(FPpart));
  hipMalloc(&d_v, nop * sizeof(FPpart));
  hipMalloc(&d_w, nop * sizeof(FPpart));
  hipMalloc(&d_XN_flat, nxn * nyn * nzn * sizeof(FPpart));
  hipMalloc(&d_YN_flat, nxn * nyn * nzn * sizeof(FPpart));
  hipMalloc(&d_ZN_flat, nxn * nyn * nzn * sizeof(FPpart));
  hipMalloc(&d_Ex_flat, nxn * nyn * nzn * sizeof(FPfield));
  hipMalloc(&d_Ey_flat, nxn * nyn * nzn * sizeof(FPfield));
  hipMalloc(&d_Ez_flat, nxn * nyn * nzn * sizeof(FPfield));
  hipMalloc(&d_Bxn_flat, nxn * nyn * nzn * sizeof(FPfield));
  hipMalloc(&d_Byn_flat, nxn * nyn * nzn * sizeof(FPfield));
  hipMalloc(&d_Bzn_flat, nxn * nyn * nzn * sizeof(FPfield));

  // Copy data from host to device
  hipMemcpy(d_x, part->x, nop * sizeof(FPpart), hipMemcpyHostToDevice);
  hipMemcpy(d_y, part->y, nop * sizeof(FPpart), hipMemcpyHostToDevice);
  hipMemcpy(d_z, part->z, nop * sizeof(FPpart), hipMemcpyHostToDevice);
  hipMemcpy(d_u, part->u, nop * sizeof(FPpart), hipMemcpyHostToDevice);
  hipMemcpy(d_v, part->v, nop * sizeof(FPpart), hipMemcpyHostToDevice);
  hipMemcpy(d_w, part->w, nop * sizeof(FPpart), hipMemcpyHostToDevice);
  hipMemcpy(d_XN_flat, grd->XN_flat, nxn * nyn * nzn * sizeof(FPpart),
             hipMemcpyHostToDevice);
  hipMemcpy(d_YN_flat, grd->YN_flat, nxn * nyn * nzn * sizeof(FPpart),
             hipMemcpyHostToDevice);
  hipMemcpy(d_ZN_flat, grd->ZN_flat, nxn * nyn * nzn * sizeof(FPpart),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Ex_flat, field->Ex_flat, nxn * nyn * nzn * sizeof(FPfield),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Ey_flat, field->Ey_flat, nxn * nyn * nzn * sizeof(FPfield),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Ez_flat, field->Ez_flat, nxn * nyn * nzn * sizeof(FPfield),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Bxn_flat, field->Bxn_flat, nxn * nyn * nzn * sizeof(FPfield),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Byn_flat, field->Byn_flat, nxn * nyn * nzn * sizeof(FPfield),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Bzn_flat, field->Bzn_flat, nxn * nyn * nzn * sizeof(FPfield),
             hipMemcpyHostToDevice);

  // Initialize number of threads
  int threadsPerBlock = 256;
  int blocksPerGrid = (part->nop + threadsPerBlock - 1) / threadsPerBlock;

  // Loop over all sub cycles. Wait untill finished for a number of points
  // before you go to the next
  for (int i_sub = 0; i_sub < part->n_sub_cycles; i_sub++) {
    mover_PC_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_x, d_y, d_z, d_u, d_v, d_w, d_XN_flat, d_YN_flat, d_ZN_flat,
        d_Ex_flat, d_Ey_flat, d_Ez_flat, d_Bxn_flat, d_Byn_flat, d_Bzn_flat,
        grd->invVOL, grd->xStart, grd->yStart, grd->zStart, grd->invdx,
        grd->invdy, grd->invdz, param->Lx, param->Ly, param->Lz, dt_sub_cycling,
        dto2, qomdt2, param->PERIODICX, param->PERIODICY, param->PERIODICZ, nxn,
        nyn, nzn, nop, part->NiterMover);
    hipDeviceSynchronize();
  }

  // Copy back to CPU
  hipMemcpy(part->x, d_x, nop * sizeof(FPpart), hipMemcpyDeviceToHost);
  hipMemcpy(part->y, d_y, nop * sizeof(FPpart), hipMemcpyDeviceToHost);
  hipMemcpy(part->z, d_z, nop * sizeof(FPpart), hipMemcpyDeviceToHost);
  hipMemcpy(part->u, d_u, nop * sizeof(FPpart), hipMemcpyDeviceToHost);
  hipMemcpy(part->v, d_v, nop * sizeof(FPpart), hipMemcpyDeviceToHost);
  hipMemcpy(part->w, d_w, nop * sizeof(FPpart), hipMemcpyDeviceToHost);

  // Free memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_XN_flat);
  hipFree(d_YN_flat);
  hipFree(d_ZN_flat);
  hipFree(d_Ex_flat);
  hipFree(d_Ey_flat);
  hipFree(d_Ez_flat);
  hipFree(d_Bxn_flat);
  hipFree(d_Byn_flat);
  hipFree(d_Bzn_flat);

  return 0;
}

/** Interpolation Particle --> Grid: This is for species */
void interpP2G(struct particles* part, struct interpDensSpecies* ids,
               struct grid* grd) {
  // arrays needed for interpolation
  FPpart weight[2][2][2];
  FPpart temp[2][2][2];
  FPpart xi[2], eta[2], zeta[2];

  // index of the cell
  int ix, iy, iz;

  for (register long long i = 0; i < part->nop; i++) {
    // determine cell: can we change to int()? is it faster?
    ix = 2 + int(floor((part->x[i] - grd->xStart) * grd->invdx));
    iy = 2 + int(floor((part->y[i] - grd->yStart) * grd->invdy));
    iz = 2 + int(floor((part->z[i] - grd->zStart) * grd->invdz));

    // distances from node
    xi[0] = part->x[i] - grd->XN[ix - 1][iy][iz];
    eta[0] = part->y[i] - grd->YN[ix][iy - 1][iz];
    zeta[0] = part->z[i] - grd->ZN[ix][iy][iz - 1];
    xi[1] = grd->XN[ix][iy][iz] - part->x[i];
    eta[1] = grd->YN[ix][iy][iz] - part->y[i];
    zeta[1] = grd->ZN[ix][iy][iz] - part->z[i];

    // calculate the weights for different nodes
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          weight[ii][jj][kk] =
              part->q[i] * xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;

    //////////////////////////
    // add charge density
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->rhon[ix - ii][iy - jj][iz - kk] +=
              weight[ii][jj][kk] * grd->invVOL;

    ////////////////////////////
    // add current density - Jx
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->u[i] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->Jx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    ////////////////////////////
    // add current density - Jy
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->v[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->Jy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    ////////////////////////////
    // add current density - Jz
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->w[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->Jz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    ////////////////////////////
    // add pressure pxx
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->u[i] * part->u[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->pxx[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    ////////////////////////////
    // add pressure pxy
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->u[i] * part->v[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->pxy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    /////////////////////////////
    // add pressure pxz
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->u[i] * part->w[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->pxz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    /////////////////////////////
    // add pressure pyy
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->v[i] * part->v[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->pyy[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    /////////////////////////////
    // add pressure pyz
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->v[i] * part->w[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->pyz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;

    /////////////////////////////
    // add pressure pzz
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          temp[ii][jj][kk] = part->w[i] * part->w[i] * weight[ii][jj][kk];
    for (int ii = 0; ii < 2; ii++)
      for (int jj = 0; jj < 2; jj++)
        for (int kk = 0; kk < 2; kk++)
          ids->pzz[ix - ii][iy - jj][iz - kk] += temp[ii][jj][kk] * grd->invVOL;
  }
}

// **************************************
//    Tot. Simulation Time (s) = 17.2564
//    Mover Time / Cycle   (s) = 0.26256
//    Interp. Time / Cycle (s) = 1.15199
// ************************************
